#include "hip/hip_runtime.h"
/*
#==============================================================================#
#--  Project Name: Network optimization using ACO                            --#

#--  Task        : RPC Implementation of Ant Colony Optimization             --#
                   - Server Side Implementation

#--  Author      : Avinash Joshi                                             --#

#--	 Date        : 23/01/13                                                  --#

#--  Version     : 1.0                                                       --#

#--  Description : Below code is the server part of the code for a remote 
                   procedure call of aco. This code needs to be compiled and 
				   executed before calling it on the client.                   
				   Also ensure that the rpcgen generated files 
				   (from remotegpuAco.x) remotegpuAco_xdr.c and 
				   remotegpuAco_svc.c are linked while compiling this code   --#
#==============================================================================#
*/

//==============
//  Header Files
//==============

#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hipblas.h>
#include <hiprand.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <rpc/rpc.h>

#include "remotegpuAco.h"


//================
//Kernel functions
//================

//Takes the inverse of the distance 
__global__ void modifyDistMatrix( float *distMatrix, int N, int nNodes){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N) {
		if(idx % nNodes == idx / nNodes){
			distMatrix[idx] = 0;		
		}
		else {
			distMatrix[idx] = 1/distMatrix[idx];
		}
	}
}

//Initialize all the pheromones to 1
__global__ void initializeMatrices( float *pherMat, int N){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N) {
		pherMat[idx] = 1;
	}
}

//Evaporates pheromones 
__global__ void evapPherMatrix( float *a, float rho, int N){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N) {
		a[idx] = (1-rho)*a[idx];
	}
}

//Make a copy so that each ant gets the same pheromone matrix
__global__ void mkCpyPherMatrix( float *a, float *b, int N){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N) {
		b[idx] = a[idx];
	}
}

//Add pheromones on the visited arcs
__global__ void addPherMatrix( float *pherMat, int *tour, float ph, float Q, int nNodes){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx<nNodes){
		pherMat[tour[idx]*nNodes + tour[idx+1]] += Q/ph;
	}	
}

//Calculate score all the arcs
__global__ void multMatrix( float *eta, float *tau, float *prob, float alpha, float beta, int N){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N) {
		prob[idx] = pow(eta[idx],beta) * pow(tau[idx],alpha);
	}
} 

//Calculate the probabilities for a move from a given node to all the connected to it
__global__ void computeProbMatrix( float *probMatrix, float sumDen, int nNodes){//Calculate probability
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < nNodes) {
		probMatrix[idx] = probMatrix[idx]/sumDen;
	}
}

//Drop the already visited nodes by making the probability 0
__global__ void dropNode( float *probMatrix, int node, int N, int nNodes ){//Drop node
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(idx < N){
		if( idx % nNodes == node || idx / nNodes == node){
			probMatrix[idx] = 0;
		}
	}
}

//==========================
//  Start of the server code
//==========================

output_data * rpcgpuaco_1(input_data *input, CLIENT *client){
	
	static output_data result;
	hipEvent_t tic, toc;
	float elapsedTime;

	hipEventCreate(&tic);
	hipEventCreate(&toc);
	hipEventRecord(tic,0);
	
//================================
//  Filling up the input arguments
//================================
	int	nNodes = input->st_nNodes, nAnts = input->st_nAnts, maxIterations = input->st_maxIterations;
	int  convergenceNo = input->st_convergenceNo;
	float rho = input->st_rho, alpha = input->st_alpha, beta = input->st_beta, Q = input->st_Q;
 
	
//======================
//  Variable Declaration
//======================
	//On Host
	float *distMat_h, *probMat_h, rowSum,  mover, graphLength = 0, bestGraphLength = 1e5, bestPrevGraphLength = 0;

	int *fullNodeList, *tour_h, *bestTour_h, N = nNodes*nNodes;

	//On device
	float *probMat_d, *pherMat_d, *curr_pherMat_d, *distMat_d;
	
	int *tour_d;
	
	int iterNo = 0, starter, counter=0;

	if(N != input->st_distMat_h.st_distMat_h_len){
		printf("Number of elements in the array is not equal to square of number nodes");
		exit(1);
	}
	
//===========================
//	Memory Allocation on host
//===========================
	distMat_h = (float *)malloc(N*sizeof(float));
	probMat_h = (float *)malloc(N*sizeof(float));
	fullNodeList = (int *)malloc(nNodes*sizeof(int));
	tour_h = (int *)malloc((nNodes + 1)*sizeof(int));
	bestTour_h = (int *)malloc((nNodes + 1)*sizeof(int));	

//=============================
//	Memory Allocation on device
//=============================		
	hipMalloc( (void **)&probMat_d, N*sizeof(float));
	hipMalloc( (void **)&pherMat_d, N*sizeof(float));
	hipMalloc( (void **)&curr_pherMat_d, N*sizeof(float));		
	hipMalloc( (void **)&distMat_d, N*sizeof(float));
	hipMalloc( (void **)&tour_d, (nNodes + 1)*sizeof(int)); 	
	
//=================
//Filling up Arrays
//=================

/*	FILE *fp;
	float ina;
	fp = fopen("input.txt","r");
	int l = 0;
	while(fscanf(fp,"%f",&ina) != EOF){
		distMat_h[l] = ina;
		l++;
	}
	fclose(fp);
*/
//	for( int i = 0; i < N; i++) distMat_h[i] = i;
	distMat_h = input->st_distMat_h.st_distMat_h_val;
	for( int i = 0; i < nNodes; i++) fullNodeList[i] = i;

//=================================
//Sending required arrays to device
//=================================

	hipMemcpy( distMat_d, distMat_h, N*sizeof(float), hipMemcpyHostToDevice);
	
//=============================
// Threads & Blocks calculation
//=============================

	int nThreads = input->st_nThreads;
	int nBlocks   = N / nThreads + ( N % nThreads == 0 ? 0 : 1 );
	
//========================
//	Start of ACO algorithm
//========================

	initializeMatrices <<< nBlocks, nThreads >>> (pherMat_d, N); // Initialize pheromone matrix to 1
	
	modifyDistMatrix <<< nBlocks, nThreads >>> (distMat_d, N, nNodes); // Invert the distances as minimization problem
	
	srand(time(NULL));
	
	while(iterNo <= maxIterations && counter <= convergenceNo){ //Iteration loop

		starter = ((float)(rand()/ (float)RAND_MAX)*nNodes);
		tour_h[0] = tour_h[nNodes] =  fullNodeList[starter];
		mkCpyPherMatrix <<< nBlocks, nThreads>>> (pherMat_d, curr_pherMat_d, N);
		
		for(int ant = 0; ant < nAnts; ant++){//Each ant loop

			multMatrix <<< nBlocks, nThreads >>> (distMat_d, curr_pherMat_d, probMat_d, alpha, beta, N);

			for(int i = 0; i < (nNodes - 1); i++){ //Graph construction loop
				
				rowSum = hipblasSasum(nNodes, probMat_d + nNodes*tour_h[i], 1);

				if(rowSum == 0) rowSum = 1;
				
				computeProbMatrix <<< nBlocks, nThreads >>> (probMat_d + nNodes*tour_h[i], rowSum, nNodes);

/*				for(int j = nNodes*tour_h[i]; j < nNodes*(tour_h[i] + 1); j++){//cumulative sum of the prob matrix for the corresponding row

					probMat_h[j] = hipblasSasum((j-nNodes*tour_h[i]) + 1, probMat_d + nNodes*tour_h[i], 1);
				}
*/
				hipMemcpy(probMat_h + nNodes*tour_h[i], probMat_d + nNodes*tour_h[i], nNodes*sizeof(float), hipMemcpyDeviceToHost);

				thrust::inclusive_scan(probMat_h + nNodes*tour_h[i], probMat_h + nNodes*(tour_h[i] + 1), probMat_h + nNodes*tour_h[i]);
				
				mover = (float)(rand()/ (float)RAND_MAX);

				for(int j = tour_h[i]*nNodes; j < (tour_h[i]*nNodes+nNodes); j++){//Next node selection
					
					if(mover < probMat_h[j]){
						tour_h[i + 1] = fullNodeList[j - (tour_h[i]*nNodes)];
						break;
					}
				}

				dropNode <<< nBlocks, nThreads >>> (probMat_d,tour_h[i], N, nNodes);
				graphLength += distMat_h[nNodes*tour_h[i] + tour_h[i + 1]];
			}//Graph construction loop

			graphLength += distMat_h[nNodes*tour_h[nNodes - 1] + tour_h[nNodes]];

			//Pheromone update
			evapPherMatrix <<< nBlocks, nThreads >>> (pherMat_d, rho, N); //Evaporation
			hipMemcpy( tour_d, tour_h, (nNodes + 1)*sizeof(int), hipMemcpyHostToDevice);
			addPherMatrix <<< nBlocks, nThreads >>> (pherMat_d, tour_d, graphLength, Q, nNodes);

			if(graphLength < bestGraphLength){
				bestGraphLength = graphLength;
				for(int k = 0; k < (nNodes + 1); k++) bestTour_h[k] = tour_h[k];
			}
			
		}//Each ant loop
		
		if(bestGraphLength == bestPrevGraphLength){
				counter = counter + 1;
		}
		else{
			bestPrevGraphLength = bestGraphLength;
			counter = 0;
		}

		graphLength = 0;
		iterNo++;
	}//Iteration Loop
			
	hipEventRecord(toc,0);
    hipEventElapsedTime(&elapsedTime,tic,toc);
	
//============================
//Filling the output variables
//============================

	result.st_bestTour_h.st_bestTour_h_len = nNodes + 1;
	result.st_bestTour_h.st_bestTour_h_val = bestTour_h;
	result.st_bestGraphLength = bestGraphLength;	
	result.st_elapsedTime = elapsedTime/1000;
	result.st_iterNo = iterNo;

	return(&result);

//============================
//Clearing up allocated memory
//============================

	free(distMat_h);	
	free(probMat_h);
	free(fullNodeList);
	free(tour_h);
	free(bestTour_h);
	
	hipFree(probMat_d);
	hipFree(pherMat_d);
	hipFree(curr_pherMat_d);
	hipFree(distMat_d);
	hipFree(tour_d);
}

output_data * rpcgpuaco_1_svc(input_data *input, struct svc_req *){
	CLIENT *client;
	return(rpcgpuaco_1(input, client));
} 
